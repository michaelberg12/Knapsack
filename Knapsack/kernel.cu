﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string>
#include <cstdint>
#include <iostream>
#include <fstream>
#include <chrono>
#include <atomic>


__global__ void knapKernel(
    const uint16_t* maxWeight, const uint16_t* arraySize,
    const uint16_t* valueD, const uint16_t* weightD,
    unsigned long long int* victorId, uint16_t* victorValue,
    unsigned long long int* offset, int* memSize, unsigned long long int runLength)
{
    unsigned long long int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int id = threadId + *offset;
    unsigned int sackWeight = 0;


    if (id <= runLength) {
        if (*offset == 0) {
            victorValue[threadId] = 0;
            victorId[threadId] = 0;
        }
        //check validity
        unsigned long long int trueIdLoc = id;
        for (int a1 = 0; trueIdLoc; a1++) {
            if (trueIdLoc & 1) {
                sackWeight += (weightD[a1]);
            }
            trueIdLoc >>= 1;
        }

        //check against memory
        unsigned int sackValue = 0;
        if (sackWeight <= *maxWeight) {
            trueIdLoc = id;
            for (int a1 = 0; trueIdLoc; a1++) {
                if (trueIdLoc & 1) {
                    sackValue += (valueD[a1]);
                }
                trueIdLoc >>= 1;
            }
            if (sackValue > victorValue[threadId]) {
                victorValue[threadId] = sackValue;
                victorId[threadId] = id;
            }
        }
    }
}

int main()
{
    uint16_t arraySize = 0;
    uint16_t maxWeight = 0;

    uint16_t* weightH = nullptr;
    uint16_t* valueH = nullptr;

    try {
        std::ifstream file("items.txt", std::ifstream::in);

        std::string line;
        for (int a1 = 0; std::getline(file, line); a1++)
        {
            switch(a1)
            {
                case 0:
                    maxWeight = (std::stoi(line));
                    break;
                case 1:
                    arraySize = std::stoi(line);
                    arraySize = static_cast<int>(arraySize);
                    weightH = (uint16_t*)malloc(sizeof(uint16_t) * arraySize);
                    valueH = (uint16_t*)malloc(sizeof(uint16_t) * arraySize);
                    break;
                default:
                    for (int a2 = 0; a2 < line.size(); a2++) {
                        if (line[a2] == ' ') {
                            std::size_t pos = line.find(" ");
                            weightH[a1 - 2] = std::stoi(line.substr(0, pos));
                            valueH[a1 - 2] = std::stoi(line.substr(pos));
                        }
                    }
            }
        }
        file.close();
    }
    catch (...) { //TODO: this is bad parctice so refactor
        std::cout << "Error reading file" << std::endl;
        return 0;
    }

    std::cout << arraySize << "\n" << maxWeight << std::endl;
    for (int a1 = 0; a1 < arraySize; a1++) {
        std::cout << "("<< weightH[a1] << " " << valueH[a1] << ")" << std::endl;
    }

    size_t size = arraySize * sizeof(uint16_t);

    hipSetDevice(0);

    uint16_t* weightD;
    hipMalloc(&weightD, size);
    uint16_t* valueD;
    hipMalloc(&valueD, size);
    uint16_t* maxWeightD;
    hipMalloc(&maxWeightD, sizeof(uint16_t));
    uint16_t* arraySizeD;
    hipMalloc(&arraySizeD, sizeof(uint16_t));
    unsigned long long int* offsetD;
    hipMalloc(&offsetD, sizeof(unsigned long long int));
    int* memSizeD;
    hipMalloc(&memSizeD, sizeof(int));

    hipMemcpy(weightD, weightH, size, hipMemcpyHostToDevice);
    hipMemcpy(valueD, valueH, size, hipMemcpyHostToDevice);
    hipMemcpy(maxWeightD, &maxWeight, sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(arraySizeD, &arraySize, sizeof(uint16_t), hipMemcpyHostToDevice);
 
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // 0-th device

    //determin block size
    int blockSize = deviceProp.maxThreadsPerMultiProcessor / deviceProp.maxBlocksPerMultiProcessor;
    unsigned long long int runLength = pow(2.0, arraySize);

    



    /// This will launch a grid that can maximally fill the GPU, on the default stream with kernel arguments
    int numBlocksPerSm = 0;
    // Number of threads my_kernel will be launched with
    int numThreads = 128;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, knapKernel, numThreads, 0);


    //determin grid size
    int memSize = numBlocksPerSm * deviceProp.multiProcessorCount * numThreads;

    uint16_t* victorValueD;
    hipMalloc(&victorValueD, sizeof(uint16_t) * memSize);
    unsigned long long int* victorIdD;
    hipMalloc(&victorIdD, sizeof(unsigned long long int) * memSize);
    //copy number of threads
    
    //copy memory size
    hipMemcpy(memSizeD, &memSize, sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(deviceProp.multiProcessorCount * numBlocksPerSm, 1, 1);

    int incriment = deviceProp.multiProcessorCount * numBlocksPerSm * numThreads;

    for (unsigned long long int offset = 0; offset < runLength; offset += incriment) {
        hipDeviceSynchronize();
        hipMemcpy(offsetD, &offset, sizeof(unsigned long long int), hipMemcpyHostToDevice);
        knapKernel <<<dimGrid, dimBlock >>> (maxWeightD, arraySizeD, valueD, weightD, victorIdD, victorValueD, offsetD, memSizeD, runLength);
    }

    //create host memory
    unsigned long long int *victorIdH = (unsigned long long int*)malloc (sizeof(unsigned long long int) * memSize);
    uint16_t *victorValueH = (uint16_t*)malloc(sizeof(uint16_t) * memSize);

    //initialize to zero
    for (int a1 = 0; a1 < memSize; a1++) {
        victorIdH[a1] = 0;
        victorValueH[a1] = 0;
    }

    hipMemcpy(victorIdH, victorIdD, sizeof(unsigned long long int) * memSize, hipMemcpyDeviceToHost);
    hipMemcpy(victorValueH, victorValueD, sizeof(uint16_t) * memSize, hipMemcpyDeviceToHost);

    //for debugging what is in memory
    /*for (int a1 = 0; a1 < memSize; a1++) {
        if (victorIdH[a1] != 0) {
            std::cout << a1 << ": " << victorIdH[a1] << ", " << victorValueH[a1] << std::endl;
        }
        
    }*/

    uint16_t highValue = 0;
    unsigned long long int highId = 0;

    for (int a1 = 0; a1 < memSize; a1++) {
        if (victorValueH[a1] > highValue) {
            highValue = victorValueH[a1];
            highId = victorIdH[a1];
        }
    }

    std::cout << "Sets to Search:" << runLength << std::endl;
    std::cout << "Mem Size:" << memSize << std::endl;
    std::cout << "Value:" << highValue << " ID:" << highId << std::endl;
    for (int a1 = 0; highId; a1 ++) {
        if (highId & 1)
            printf("%d ", a1 + 1);

        highId >>= 1;
    }
    
    //free memory
    hipFree(weightD);
    hipFree(valueD);
    hipFree(maxWeightD);
    hipFree(arraySizeD);
    hipFree(offsetD);

    free(victorIdH);
    free(victorValueH);


    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    //code from an example tutorial
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    //end code from an example tutorial

    return 0;
}


