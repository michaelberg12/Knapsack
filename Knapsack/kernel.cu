﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string>
#include <cstdint>
#include <iostream>
#include <fstream>

__global__ void knapKernel(
    const uint16_t* maxWeight, const uint16_t* arraySize, 
    const uint16_t* valueD, const uint16_t* weightD, 
    unsigned long long int* victorId, uint16_t* victorValue, 
    unsigned long long int* offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long int trueId = id + *offset;
    unsigned int sackWeight = 0;
    
    //check validity
    for (int a1 = 0; trueId; a1++) {
        if (trueId & 1) {
            sackWeight += (weightD[a1]);
        }
        trueId >>= 1;
    }
    //check against memory
    trueId = id + *offset;

    unsigned int sackValue = 0;
    if (sackWeight <= *maxWeight ) {
        for (int a1 = 0; trueId; a1++) {
            if (trueId & 1) {
                sackValue += (valueD[a1]);
            }
            trueId >>= 1;
        }
        if (sackValue > victorValue[id]) {
            victorValue[id] = sackValue;
            victorId[id] = id + *offset;
        }
    }
}

int main()
{

    uint16_t arraySize = 0;
    uint16_t maxWeight = 0;

    uint16_t* weightH = nullptr;
    uint16_t* valueH = nullptr;

    try {
        std::ifstream file("items.txt", std::ifstream::in);

        std::string line;
        for (int a1 = 0; std::getline(file, line); a1++)
        {
            switch(a1)
            {
                case 0:
                    maxWeight = (std::stoi(line));
                    break;
                case 1:
                    arraySize = std::stoi(line);
                    arraySize = static_cast<int>(arraySize);
                    weightH = (uint16_t*)malloc(sizeof(uint16_t) * arraySize);
                    valueH = (uint16_t*)malloc(sizeof(uint16_t) * arraySize);
                    break;
                default:
                    for (int a2 = 0; a2 < line.size(); a2++) {
                        if (line[a2] == ' ') {
                            std::size_t pos = line.find(" ");
                            weightH[a1 - 2] = std::stoi(line.substr(0, pos));
                            valueH[a1 - 2] = std::stoi(line.substr(pos));
                        }
                    }
            }
        }
        file.close();
    }
    catch (...) { //TODO: this is bad parctice so refactor
        std::cout << "Error reading file" << std::endl;
        return 0;
    }

    std::cout << arraySize << "\n" << maxWeight << std::endl;
    for (int a1 = 0; a1 < arraySize; a1++) {
        std::cout << "("<< weightH[a1] << " " << valueH[a1] << ")" << std::endl;
    }

    size_t size = arraySize * sizeof(uint16_t);

    hipSetDevice(0);

    uint16_t* weightD;
    hipMalloc(&weightD, size);
    uint16_t* valueD;
    hipMalloc(&valueD, size);
    uint16_t* maxWeightD;
    hipMalloc(&maxWeightD, sizeof(uint16_t));
    uint16_t* arraySizeD;
    hipMalloc(&arraySizeD, sizeof(uint16_t));
    unsigned long long int* offsetD;
    hipMalloc(&offsetD, sizeof(unsigned long long int));

    hipMemcpy(weightD, weightH, size, hipMemcpyHostToDevice);
    hipMemcpy(valueD, valueH, size, hipMemcpyHostToDevice);
    hipMemcpy(maxWeightD, &maxWeight, sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(arraySizeD, &arraySize, sizeof(uint16_t), hipMemcpyHostToDevice);
 
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // 0-th device

    int blockSize = deviceProp.maxThreadsPerMultiProcessor / deviceProp.maxBlocksPerMultiProcessor;
    unsigned long long int numThreads = pow(2.0, arraySize);
    int numBlocks = deviceProp.maxBlocksPerMultiProcessor * deviceProp.multiProcessorCount;

    uint16_t* victorValueD;
    hipMalloc(&victorValueD, sizeof(uint16_t) * numBlocks * deviceProp.maxThreadsPerBlock);
    unsigned long long int* victorIdD;
    hipMalloc(&victorIdD, sizeof(unsigned long long int) * numBlocks * deviceProp.maxThreadsPerBlock);

    //capable up to n=50 15 minutes run time 
    //the cpu finish the computation meaning that under ~ n=21 all computation is done by the cpu
    std::cout << numThreads << std::endl;
    for (unsigned long long int a1 = 0; a1 < numThreads; a1 += numBlocks * deviceProp.maxThreadsPerBlock) {
        
        hipMemcpy(offsetD, &a1, sizeof(unsigned long long int), hipMemcpyHostToDevice);
        knapKernel <<<numBlocks, blockSize >>> (maxWeightD, arraySizeD, valueD, weightD, victorIdD, victorValueD, offsetD);
    }
    
    //create host memory
    unsigned long long int*victorIdH = (unsigned long long int*)malloc (sizeof(unsigned long long int) * (numBlocks * deviceProp.maxThreadsPerBlock));
    uint16_t *victorValueH = (uint16_t*)malloc(sizeof(uint16_t) * (numBlocks * deviceProp.maxThreadsPerBlock));

    hipMemcpy(victorIdH   , victorIdD,    sizeof(unsigned long long int) * numBlocks * deviceProp.maxThreadsPerBlock, hipMemcpyDeviceToHost);
    hipMemcpy(victorValueH, victorValueD, sizeof(uint16_t) * numBlocks * deviceProp.maxThreadsPerBlock, hipMemcpyDeviceToHost);

    int highValue = 0;
    unsigned long long int highId = 0;

    for (int a1 = 0; a1 < numBlocks * deviceProp.maxThreadsPerBlock; a1++) {
        if (victorValueH[a1] > highValue) {
            highValue = victorValueH[a1];
            highId = victorIdH[a1];
        }
    }

    std::cout << "Value:" << highValue << " ID:" << highId << std::endl;
    for (int a1 = 0; highId; a1 ++) {
        if (highId & 1)
            printf("%d ", a1 + 1);

        highId >>= 1;
    }
    
    //free memory
    hipFree(weightD);
    hipFree(valueD);
    hipFree(maxWeightD);
    hipFree(arraySizeD);
    hipFree(offsetD);

    //delete victorIdH;
    //delete victorValueH;


    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    //code from an example tutorial
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    //end code from an example tutorial

    return 0;
}


